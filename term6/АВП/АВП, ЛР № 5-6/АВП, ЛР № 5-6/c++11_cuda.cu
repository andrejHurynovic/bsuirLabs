#include "hip/hip_runtime.h"

#include <hip/device_functions.h>
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <chrono>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define CHANNELS 3
#define WINDOW_SIZE 3
#define TILE_SIZE 15

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n %s",__FILE__,__LINE__, hipGetErrorString(x));\
    exit(EXIT_FAILURE);}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    exit(EXIT_FAILURE);}} while(0)

typedef std::chrono::high_resolution_clock ttime;
typedef ttime::time_point time_point;
#define DURATION(x) std::chrono::duration_cast<std::chrono::microseconds>(ttime::now() - x).count()


__device__ __inline__ void Sort(int& a, int& b)
{
	const int d = a - b;
	const int m = ~(d >> 8);
	b += d & m;
	a -= d & m;
}

__global__ void medianFilterSharedKernel(unsigned char* inputImageKernel, unsigned char* outputImagekernel, int imageWidth, int imageHeight)
{
	//Set the row and col value for each thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("\n%d", row);
	__shared__ unsigned char sharedmem[(TILE_SIZE + 2)][(TILE_SIZE + 2)];  //initialize shared memory
	//Take some values.
	bool is_x_left = (threadIdx.x == 0), is_x_right = (threadIdx.x == TILE_SIZE - 1);
	bool is_y_top = (threadIdx.y == 0), is_y_bottom = (threadIdx.y == TILE_SIZE - 1);

	//Initialize with zero
	if (is_x_left)
		sharedmem[threadIdx.x][threadIdx.y + 1] = 0;
	else if (is_x_right)
		sharedmem[threadIdx.x + 2][threadIdx.y + 1] = 0;
	if (is_y_top) {
		sharedmem[threadIdx.x + 1][threadIdx.y] = 0;
		if (is_x_left)
			sharedmem[threadIdx.x][threadIdx.y] = 0;
		else if (is_x_right)
			sharedmem[threadIdx.x + 2][threadIdx.y] = 0;
	}
	else if (is_y_bottom) {
		sharedmem[threadIdx.x + 1][threadIdx.y + 2] = 0;
		if (is_x_right)
			sharedmem[threadIdx.x + 2][threadIdx.y + 2] = 0;
		else if (is_x_left)
			sharedmem[threadIdx.x][threadIdx.y + 2] = 0;
	}

	//Setup pixel values
	sharedmem[threadIdx.x + 1][threadIdx.y + 1] = inputImageKernel[row * imageWidth + col];
	//Check for boundry conditions.
	if (is_x_left && (col > 0))
		sharedmem[threadIdx.x][threadIdx.y + 1] = inputImageKernel[row * imageWidth + (col - 1)];
	else if (is_x_right && (col < imageWidth - 1))
		sharedmem[threadIdx.x + 2][threadIdx.y + 1] = inputImageKernel[row * imageWidth + (col + 1)];
	if (is_y_top && (row > 0)) {
		sharedmem[threadIdx.x + 1][threadIdx.y] = inputImageKernel[(row - 1) * imageWidth + col];
		if (is_x_left)
			sharedmem[threadIdx.x][threadIdx.y] = inputImageKernel[(row - 1) * imageWidth + (col - 1)];
		else if (is_x_right)
			sharedmem[threadIdx.x + 2][threadIdx.y] = inputImageKernel[(row - 1) * imageWidth + (col + 1)];
	}
	else if (is_y_bottom && (row < imageHeight - 1)) {
		sharedmem[threadIdx.x + 1][threadIdx.y + 2] = inputImageKernel[(row + 1) * imageWidth + col];
		if (is_x_right)
			sharedmem[threadIdx.x + 2][threadIdx.y + 2] = inputImageKernel[(row + 1) * imageWidth + (col + 1)];
		else if (is_x_left)
			sharedmem[threadIdx.x][threadIdx.y + 2] = inputImageKernel[(row + 1) * imageWidth + (col - 1)];
	}

	__syncthreads();   //Wait for all threads to be done.

	//Setup the filter.
	int a[9] = { sharedmem[threadIdx.x][threadIdx.y], sharedmem[threadIdx.x + 1][threadIdx.y], sharedmem[threadIdx.x + 2][threadIdx.y],
				  sharedmem[threadIdx.x][threadIdx.y + 1], sharedmem[threadIdx.x + 1][threadIdx.y + 1], sharedmem[threadIdx.x + 2][threadIdx.y + 1],
				  sharedmem[threadIdx.x][threadIdx.y + 2], sharedmem[threadIdx.x + 1][threadIdx.y + 2], sharedmem[threadIdx.x + 2][threadIdx.y + 2] };

	Sort(a[1], a[2]); Sort(a[4], a[5]); Sort(a[7], a[8]);
	Sort(a[0], a[1]); Sort(a[3], a[4]); Sort(a[6], a[7]);
	Sort(a[1], a[2]); Sort(a[4], a[5]); Sort(a[7], a[8]);
	Sort(a[0], a[3]); Sort(a[5], a[8]); Sort(a[4], a[7]);
	Sort(a[3], a[6]); Sort(a[1], a[4]); Sort(a[2], a[5]);
	Sort(a[4], a[7]); Sort(a[4], a[2]); Sort(a[6], a[4]);
	Sort(a[4], a[2]);

	outputImagekernel[row * imageWidth + col] = (uint8_t)a[4];   //Set the output image values.
}

__global__ void rgbMedianSharedFilter(unsigned char* inputImageKernel, unsigned char* outputImagekernel, int imageWidth, int imageHeight)
{
	//Set the row and col value for each thread.
	int threadX = threadIdx.x * 3;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ unsigned char sharedmem[(TILE_SIZE + 2) * 3][(TILE_SIZE + 2)];  //initialize shared memory
	//Take some values.
	bool is_x_left = (threadIdx.x == 0), is_x_right = (threadIdx.x == TILE_SIZE - 1);
	bool is_y_top = (threadIdx.y == 0), is_y_bottom = (threadIdx.y == TILE_SIZE - 1);

	//Initialize with zero
	if (is_x_left) {
		sharedmem[threadIdx.x][threadIdx.y + 1] = 0;
		sharedmem[threadIdx.x + 1][threadIdx.y + 1] = 0;
		sharedmem[threadIdx.x + 2][threadIdx.y + 1] = 0;
	}
	else if (is_x_right) {
		sharedmem[threadX + 3][threadIdx.y + 1] = 0;
		sharedmem[threadX + 4][threadIdx.y + 1] = 0;
		sharedmem[threadX + 5][threadIdx.y + 1] = 0;
	}

	if (is_y_top) {
		sharedmem[threadX][threadIdx.y] = 0;
		sharedmem[threadX + 1][threadIdx.y] = 0;
		sharedmem[threadX + 2][threadIdx.y] = 0;
		if (is_x_left) {
			sharedmem[threadIdx.x][threadIdx.y] = 0;
			sharedmem[threadIdx.x + 1][threadIdx.y] = 0;
			sharedmem[threadIdx.x + 2][threadIdx.y] = 0;
		}
		else if (is_x_right) {
			sharedmem[threadX + 3][threadIdx.y] = 0;
			sharedmem[threadX + 4][threadIdx.y] = 0;
			sharedmem[threadX + 5][threadIdx.y] = 0;
		}
	}
	else if (is_y_bottom) {
		sharedmem[threadX][threadIdx.y + 2] = 0;
		sharedmem[threadX + 1][threadIdx.y + 2] = 0;
		sharedmem[threadX + 2][threadIdx.y + 2] = 0;
		if (is_x_right) {
			sharedmem[threadX + 3][threadIdx.y + 2] = 0;
			sharedmem[threadX + 4][threadIdx.y + 2] = 0;
			sharedmem[threadX + 5][threadIdx.y + 2] = 0;
		}
		else if (is_x_left) {
			sharedmem[threadIdx.x][threadIdx.y + 2] = 0;
			sharedmem[threadIdx.x + 1][threadIdx.y + 2] = 0;
			sharedmem[threadIdx.x + 2][threadIdx.y + 2] = 0;
		}
	}

	//Setup pixel values
	sharedmem[threadIdx.x + 1][threadIdx.y + 1] = inputImageKernel[row * imageWidth + col];
	//Check for boundry conditions.
	if (is_x_left && (col > 0))
		sharedmem[threadIdx.x][threadIdx.y + 1] = inputImageKernel[row * imageWidth + (col - 1)];
	else if (is_x_right && (col < imageWidth - 1))
		sharedmem[threadIdx.x + 2][threadIdx.y + 1] = inputImageKernel[row * imageWidth + (col + 1)];
	if (is_y_top && (row > 0)) {
		sharedmem[threadIdx.x + 1][threadIdx.y] = inputImageKernel[(row - 1) * imageWidth + col];
		if (is_x_left)
			sharedmem[threadIdx.x][threadIdx.y] = inputImageKernel[(row - 1) * imageWidth + (col - 1)];
		else if (is_x_right)
			sharedmem[threadIdx.x + 2][threadIdx.y] = inputImageKernel[(row - 1) * imageWidth + (col + 1)];
	}
	else if (is_y_bottom && (row < imageHeight - 1)) {
		sharedmem[threadIdx.x + 1][threadIdx.y + 2] = inputImageKernel[(row + 1) * imageWidth + col];
		if (is_x_right)
			sharedmem[threadIdx.x + 2][threadIdx.y + 2] = inputImageKernel[(row + 1) * imageWidth + (col + 1)];
		else if (is_x_left)
			sharedmem[threadIdx.x][threadIdx.y + 2] = inputImageKernel[(row + 1) * imageWidth + (col - 1)];
	}

	__syncthreads();   //Wait for all threads to be done.

	//Setup the filter.
	int a[9] = { sharedmem[threadIdx.x][threadIdx.y], sharedmem[threadIdx.x + 1][threadIdx.y], sharedmem[threadIdx.x + 2][threadIdx.y],
				  sharedmem[threadIdx.x][threadIdx.y + 1], sharedmem[threadIdx.x + 1][threadIdx.y + 1], sharedmem[threadIdx.x + 2][threadIdx.y + 1],
				  sharedmem[threadIdx.x][threadIdx.y + 2], sharedmem[threadIdx.x + 1][threadIdx.y + 2], sharedmem[threadIdx.x + 2][threadIdx.y + 2] };

	Sort(a[1], a[2]); Sort(a[4], a[5]); Sort(a[7], a[8]);
	Sort(a[0], a[1]); Sort(a[3], a[4]); Sort(a[6], a[7]);
	Sort(a[1], a[2]); Sort(a[4], a[5]); Sort(a[7], a[8]);
	Sort(a[0], a[3]); Sort(a[5], a[8]); Sort(a[4], a[7]);
	Sort(a[3], a[6]); Sort(a[1], a[4]); Sort(a[2], a[5]);
	Sort(a[4], a[7]); Sort(a[4], a[2]); Sort(a[6], a[4]);
	Sort(a[4], a[2]);

	outputImagekernel[row * imageWidth + col] = (uint8_t)a[4];   //Set the output image values.
}

__global__ void mFilter(const uint8_t* in, uint8_t* out, const size_t W, const size_t H)
{
	//int index = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int idy = threadIdx.y + blockIdx.y * blockDim.y + 1;
	int a[9] = {};
	if (idy < H - 1 && idx < W - 1)
	{
		for (int z2 = 0; z2 < 3; ++z2)
			for (int z1 = 0; z1 < 3; ++z1)
				a[3 * z2 + z1] = in[(idy - 1 + z2) * W + idx - 1 + z1];

		Sort(a[1], a[2]); Sort(a[4], a[5]); Sort(a[7], a[8]);
		Sort(a[0], a[1]); Sort(a[3], a[4]); Sort(a[6], a[7]);
		Sort(a[1], a[2]); Sort(a[4], a[5]); Sort(a[7], a[8]);
		Sort(a[0], a[3]); Sort(a[5], a[8]); Sort(a[4], a[7]);
		Sort(a[3], a[6]); Sort(a[1], a[4]); Sort(a[2], a[5]);
		Sort(a[4], a[7]); Sort(a[4], a[2]); Sort(a[6], a[4]);
		Sort(a[4], a[2]);

		out[idy * W + idx] = (uint8_t)a[4];
	}
}

__global__ void rgbMedianFilterGPU(const uint8_t* in, uint8_t* out, const size_t W, const size_t H) {
	//int index = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = threadIdx.x * 3 + blockIdx.x * blockDim.x + 3;
	int idy = threadIdx.y + blockIdx.y * blockDim.y + 1;
	//int idy = 4;
	int r[9] = {};
	int g[9] = {};
	int b[9] = {};
	if (idy < H - 1 && idx < W - 3)
	{
		for (int z2 = 0; z2 < 3; ++z2)
			for (int z1 = 0; z1 < 3; ++z1) {
				r[z2 * 3 + z1] = in[(idy - 1 + z2) * W + idx - 3 + z1 * 3];
				g[z2 * 3 + z1] = in[(idy - 1 + z2) * W + idx - 3 + z1 * 3 + 1];
				b[z2 * 3 + z1] = in[(idy - 1 + z2) * W + idx - 3 + z1 * 3 + 2];
			}

		Sort(r[1], r[2]); Sort(r[4], r[5]); Sort(r[7], r[8]);
		Sort(r[0], r[1]); Sort(r[3], r[4]); Sort(r[6], r[7]);
		Sort(r[1], r[2]); Sort(r[4], r[5]); Sort(r[7], r[8]);
		Sort(r[0], r[3]); Sort(r[5], r[8]); Sort(r[4], r[7]);
		Sort(r[3], r[6]); Sort(r[1], r[4]); Sort(r[2], r[5]);
		Sort(r[4], r[7]); Sort(r[4], r[2]); Sort(r[6], r[4]);
		Sort(r[4], r[2]);

		Sort(g[1], g[2]); Sort(g[4], g[5]); Sort(g[7], g[8]);
		Sort(g[0], g[1]); Sort(g[3], g[4]); Sort(g[6], g[7]);
		Sort(g[1], g[2]); Sort(g[4], g[5]); Sort(g[7], g[8]);
		Sort(g[0], g[3]); Sort(g[5], g[8]); Sort(g[4], g[7]);
		Sort(g[3], g[6]); Sort(g[1], g[4]); Sort(g[2], g[5]);
		Sort(g[4], g[7]); Sort(g[4], g[2]); Sort(g[6], g[4]);
		Sort(g[4], g[2]);

		Sort(b[1], b[2]); Sort(b[4], b[5]); Sort(b[7], b[8]);
		Sort(b[0], b[1]); Sort(b[3], b[4]); Sort(b[6], b[7]);
		Sort(b[1], b[2]); Sort(b[4], b[5]); Sort(b[7], b[8]);
		Sort(b[0], b[3]); Sort(b[5], b[8]); Sort(b[4], b[7]);
		Sort(b[3], b[6]); Sort(b[1], b[4]); Sort(b[2], b[5]);
		Sort(b[4], b[7]); Sort(b[4], b[2]); Sort(b[6], b[4]);
		Sort(b[4], b[2]);

		out[idy * W + idx] = (uint8_t)r[4];
		out[idy * W + idx + 1] = (uint8_t)g[4];
		out[idy * W + idx + 2] = (uint8_t)b[4];
	}
}

bool MedianFilterGPU(uint8_t* image, uint8_t* outputImage, const size_t width, const size_t height, bool useSharedMemory) {
	//Cuda error and image values.
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	int size = width * height * sizeof(char);

	//Initialize images.
	unsigned char* inputDeviceImage;
	CUDA_CALL(hipMalloc((void**)&inputDeviceImage, size));
	CUDA_CALL(hipMemcpy(inputDeviceImage, image, size, hipMemcpyHostToDevice));
	
	uint8_t* deviceOutputImage;
	hipMalloc((void**)&deviceOutputImage, size);
	//take block and grids.
	dim3 dimBlock(TILE_SIZE, TILE_SIZE);
	dim3 dimGrid((int)ceil((float)width / (float)TILE_SIZE), (int)ceil((float)height / (float)TILE_SIZE));

	//printf("\nDim grid %d %d %d\n", dimGrid.x, dimGrid.y, dimGrid.x * dimGrid.y);

	struct hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	size_t maxThreads = prop.maxGridSize[0];

	//Check for shared memories and call the kernel
	if (!useSharedMemory) {
		if (CHANNELS == 1) {
			mFilter << <dimGrid, dimBlock >> > (inputDeviceImage, deviceOutputImage, width, height);
		}
		else {
			rgbMedianFilterGPU << <dimGrid, dimBlock >> > (inputDeviceImage, deviceOutputImage, width, height);
		}
	} else {
		dimGrid = ((int)floorf((float)width / (float)TILE_SIZE), (int)floorf((float)height / (float)TILE_SIZE));
		if (CHANNELS == 1) {
			medianFilterSharedKernel << <dimGrid, dimBlock >> > (inputDeviceImage, deviceOutputImage, width, height);
		}
		else {
			rgbMedianSharedFilter << <dimGrid, dimBlock >> > (inputDeviceImage, deviceOutputImage, width, height);
		}
	}


	//CUDA_CALL(hipMalloc((void**)&deviceOutputImage, size));
	//CUDA_CALL(hipMalloc())

	// save output image to host.
	CUDA_CALL(hipMemcpy(outputImage, deviceOutputImage, size, hipMemcpyDeviceToHost));

	//Free the memory
	hipFree(inputDeviceImage);
	hipFree(deviceOutputImage);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float time = 0;
	hipEventElapsedTime(&time, start, stop);
	printf("Time: %f ms\n", time);
	return true;
}

void Sort_CPU(int& a, int& b)
{
	const int d = a - b;
	const int m = ~(d >> 8);
	b += d & m;
	a -= d & m;
}

inline void Sort_CPU(int a[9]) //частично сортирует весь массив
{
	Sort_CPU(a[1], a[2]); Sort_CPU(a[4], a[5]); Sort_CPU(a[7], a[8]);
	Sort_CPU(a[0], a[1]); Sort_CPU(a[3], a[4]); Sort_CPU(a[6], a[7]);
	Sort_CPU(a[1], a[2]); Sort_CPU(a[4], a[5]); Sort_CPU(a[7], a[8]);
	Sort_CPU(a[0], a[3]); Sort_CPU(a[5], a[8]); Sort_CPU(a[4], a[7]);
	Sort_CPU(a[3], a[6]); Sort_CPU(a[1], a[4]); Sort_CPU(a[2], a[5]);
	Sort_CPU(a[4], a[7]); Sort_CPU(a[4], a[2]); Sort_CPU(a[6], a[4]);
	Sort_CPU(a[4], a[2]);
}

void MedianFilter(const uint8_t* src, uint8_t* dst, size_t width, size_t height)
{
	time_point t = ttime::now();
	int a[9];
	for (size_t y = 1; y < height - 1; y++)
	{
		for (size_t x = 1; x < width - 1; x++)
		{
			for (int z1 = 0; z1 < 3; z1++)
				for (int z2 = 0; z2 < 3; z2++)
					a[z1 * 3 + z2] = src[(y - 1 + z1) * width + x - 1 + z2];
			Sort_CPU(a);
			dst[y * width + x] = (uint8_t)a[4];
		}
	}
	printf("Time: %f ms\n", DURATION(t) / 1000.0);
}

void MedianFilterRGB(const uint8_t* src, uint8_t* dst, size_t width, size_t height)
{
	time_point t = ttime::now();
	int r[9], g[9], b[9]; 
	for (size_t y = 1; y < height - 1; y++)
	{
		for (size_t x = 3; x < width - 3; x += 3)
		{
			for (int z1 = 0; z1 < 3; z1++)
				for (int z2 = 0; z2 < 3; z2++) {
					r[z1 * 3 + z2] = src[(y - 1 + z1) * width + x - 3 + z2 * 3 + 0];
					g[z1 * 3 + z2] = src[(y - 1 + z1) * width + x - 3 + z2 * 3 + 1];
					b[z1 * 3 + z2] = src[(y - 1 + z1) * width + x - 3 + z2 * 3 + 2];
				}
			Sort_CPU(r);
			Sort_CPU(g);
			Sort_CPU(b);
			dst[y * width + x] = (uint8_t)r[4];
			dst[y * width + x + 1] = (uint8_t)g[4];
			dst[y * width + x + 2] = (uint8_t)b[4];
		}
	}
	printf("Duration %f\n", DURATION(t) / 1000.0);
}

bool cmp(const uint8_t* p1, const uint8_t* p2, size_t size) {
	for (int i = 0; i < size; i++)
		if (p1[i] != p2[i])
			return false;
	return true;
}

int main()
{
	int width, height, bpp;

	uint8_t* image = stbi_load("input/noised.jpg", &width, &height, &bpp, CHANNELS);
	uint8_t* gpu_image = (uint8_t*)malloc(width * height * CHANNELS);
	uint8_t* cpu_image = (uint8_t*)malloc(width * height * CHANNELS);

	stbi_write_jpg("input_image.jpg", width, height, CHANNELS, image, width);

	printf("Size of image:	pixels:	%d \n		width:	%d \n		height:	%d\n", width * height, width, height);

	printf("CPU\n");
	if (CHANNELS == 1) {
		MedianFilter(image, cpu_image, width * CHANNELS, height);
	}
	else {
		MedianFilterRGB(image, cpu_image, width * CHANNELS, height);
	}

	printf("Shared memory:\n");
	MedianFilterGPU(image, gpu_image, width * CHANNELS, height, true);

	printf("Global memory:\n");
	MedianFilterGPU(image, gpu_image, width * CHANNELS, height, false);

	if (cmp(cpu_image, gpu_image, width * height * CHANNELS)) {
		printf("CPU and GPU images are same!\n");
	} else {
		printf("CPU and GPU images are't same...\n");
	}
	stbi_write_jpg("output/result_image.jpg", width, height, CHANNELS, cpu_image, width);

	stbi_image_free(image);
	stbi_image_free(gpu_image);
	stbi_image_free(cpu_image);

	return 0;
}
