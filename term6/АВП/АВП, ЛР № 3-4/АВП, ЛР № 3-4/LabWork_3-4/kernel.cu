﻿
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <chrono>
#include <ctime>

using namespace std;

void cpu_function(int* first, int* second, int n, int m) {
    for (int temp = 0; temp < (m - m % 2) * (n - n % 2); temp++) {

        
        int i = temp / ((n / 2));
        int j = temp % ((n / 2));

        if (temp > (m - m % 2) * (n - n % 2)) continue;

        if (i % 4 == 0) {
            second[i * n / 2 + j] = first[(i / 2) * n + n + j * 2 + 1];
        }

        if (i % 4 == 1) {
            second[i * n / 2 + j] = first[(i / 2) * n + n + j * 2];
        }

        if (i % 4 == 2) {
            second[i * n / 2 + j] = first[(i / 4) * (n * 2) + j * 2 + 1];
        }

        if (i % 4 == 3) {
            second[i * n / 2 + j] = first[(i / 4) * (n * 2) + j * 2];
        }

    }
}


__global__ void gpu_function(int* first, int* second, int num_thread, int n, int m) {

    int temp = threadIdx.x + blockIdx.x * num_thread;

    int i = temp / ((n / 2));
    int j = temp % ((n / 2));

    if (temp > (m - m % 2) * (n - n % 2));

    else {
        if (i % 4 == 0) {
            second[i * n / 2 + j] = first[(i / 2) * n + n + j * 2 + 1];
        }
        if (i % 4 == 1) {
            second[i * n / 2 + j] = first[(i / 2) * n + n + j * 2];
        }
        if (i % 4 == 2) {
            second[i * n / 2 + j] = first[(i / 4) * (n * 2) + j * 2 + 1];
        }
        if (i % 4 == 3) {
            second[i * n / 2 + j] = first[(i / 4) * (n * 2) + j * 2];
        }
    }
}



int main(void) {

    int n = 10000;
    int m = 10000;
    int lenght = n * m;

    int* first;
    int* second;
    int* check_second;

    int* d_first;
    int* d_second;

    first = (int*)malloc(lenght * sizeof(int));
    second = (int*)malloc(lenght * sizeof(int));
    check_second = (int*)malloc(lenght * sizeof(int));

    for (int i = 0; i < lenght; i++) {
        first[i] = i;
    }

    for (int i = 0; i < lenght; i++) {
        second[i] = -1;
    }

    for (int i = 0; i < lenght; i++) {
        check_second[i] = -1;
    }

    cout << "\n";

    dim3 blocks((lenght / 32), 1);

    hipMalloc((int**)&d_first, lenght * sizeof(int));
    hipMalloc((int**)&d_second, lenght * sizeof(int));

    hipMemcpy(d_first, first, lenght * sizeof(int), hipMemcpyHostToDevice);

    hipStream_t stream0;
    hipStreamCreate(&stream0);

    hipEvent_t e_start, e_stop;
    hipEventCreate(&e_start);
    hipEventCreate(&e_stop);
    hipEventRecord(e_start, 0);


    gpu_function <<< blocks, 32, 0, stream0 >>> (d_first, d_second, 32, n, m);

    hipDeviceSynchronize();
    hipStreamSynchronize(stream0);

    hipEventRecord(e_stop, 0);
    hipEventSynchronize(e_stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, e_start, e_stop);

    cout << "GPU time: " << elapsedTime << "ms" << endl;

 
    hipMemcpy(check_second, d_second, lenght * sizeof(int), hipMemcpyDeviceToHost);
    

    using namespace std::chrono;
    auto start = high_resolution_clock::now();

    cpu_function(first, second, n, m);

    auto stop = high_resolution_clock::now();
    duration<double> time = stop - start;
    cout << "CPU time: " << time.count() * 1000 << "ms" << endl;


    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            cout.fill(' ');
            cout.width(3);
            cout << setw(5) << check_second[i * n / 2 + j] << "  ";
        }
        cout << "\n";
    }
    cout << "\n";

    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            cout.fill(' ');
            cout.width(3);
            cout << setw(5) << second[i * n / 2 + j] << "  ";
        }
        cout << "\n";
    }
    cout << "\n";


    for (int i = 0; i < (m / 2) * 4; i++) {
        for (int j = 0; j < n / 2; j++) {
            if (second[i * n / 2 + j] != check_second[i * n / 2 + j]) {
                cout << "\nfalse\n" << i << ' ' << j;
                //return;
            }
        }
    }


    cout << "\ntrue\n";
}